#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMulKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 1024;
    size_t size = N * N * sizeof(float);

    // Allocate memory on the host
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize matrices A and B with random values
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (float)i;
        h_B[i] = (float)i;
    }

    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define the grid and block sizes
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    // Copy the result matrix from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Check the result (for demonstration purposes)
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("C[%d][%d] = %f\n", i, j, h_C[i * N + j]);
        }
    }

    return 0;
}