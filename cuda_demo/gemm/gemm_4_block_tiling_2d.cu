#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "gemm.cuh"

const int TN = 4;
const int TM = 4;
const int TILE_SIZE = 32;
const int TILE_SIZE_X = 64;
const int TILE_SIZE_Y = 64;
// 用于测速
const int REPEAT_TIME = 1000;

template <>
__global__ void smem_gemm_blocktiling_2d_kernel(const float* A, const float* B,
                                                float* C, const int M,
                                                const int N, const int K) {
  // Shared memory allocation
  // threadIdx.x一个处理TM个数据
  __shared__ float shared_A[TILE_SIZE_Y][TILE_SIZE_X];
  __shared__ float shared_B[TILE_SIZE_Y][TILE_SIZE_X];

  // Calculate row and column index
  int row = TILE_SIZE_Y * blockIdx.y + TM * threadIdx.y;
  int col = TILE_SIZE_X * blockIdx.x + TN * threadIdx.x;

  int t_row = threadIdx.y * TM;
  int t_col = threadIdx.x * TN;

  float value[TM * TN] = {0.0};
  // float tmp_a[TN] = {0.0};
  float tmp_a[TM] = {0.0};
  float tmp_b[TN] = {0.0};

  for (int t = 0; t < (K + TILE_SIZE_X - 1) / TILE_SIZE_X; ++t) {
    // Load data into shared memory
    if (row < M && t * TILE_SIZE_X + t_col < K) {
#pragma unroll
      for (int m = 0; m < TM; m++) {
        for (int n = 0; n < TN; n++) {
          shared_A[t_row + m][t_col + n] =
              A[(row + m) * K + t * TILE_SIZE_X + t_col + n];
        }
      }
    } else
      shared_A[threadIdx.y][threadIdx.x] = 0.0f;

    if (col < N && t * TILE_SIZE_Y + threadIdx.y < K) {
#pragma unroll
      for (int m = 0; m < TM; m++) {
        for (int n = 0; n < TN; n++) {
          shared_B[t_row + m][t_col + n] =
              B[(t * TILE_SIZE_Y + t_row + m) * N + col + n];
        }
      }
    } else
      shared_B[threadIdx.y][threadIdx.x] = 0.0f;

    __syncthreads();
    // Perform computation
    for (int i = 0; i < TILE_SIZE_X; i++) {
#pragma unroll
      for (int n = 0; n < TN; n++) {
        tmp_b[n] = shared_B[i][t_col + n];
      }
#pragma unroll
      for (int m = 0; m < TM; m++) {
        tmp_a[m] = shared_A[t_row + m][i];
      }

#pragma unroll
      for (int m = 0; m < TM; m++) {
#pragma unroll
        for (int n = 0; n < TN; n++) {
          value[m * TN + n] += tmp_a[m] * tmp_b[n];
        }
      }
    }
    __syncthreads();
  }

  // Write result
  if (row < M && col < N) {
#pragma unroll
    for (int m = 0; m < TM; m++) {
#pragma unroll
      for (int n = 0; n < TN; n++) {
        C[(row + m) * N + col + n] = value[m * TN + n];
      }
    }
  }
}


torch::Tensor smem_gemm_blocktiling_2d(torch::Tensor A, torch::Tensor B) {
  const int block_size_x = TILE_SIZE_X;
  const int block_size_y = TILE_SIZE_Y;
  const int M = A.size(0);
  const int K = A.size(1);
  const int N = B.size(1);

  auto C = torch::zeros({M, N}, torch::kCUDA);  // 确保在 CUDA 设备上分配内存
  dim3 grid_dim((M + block_size_x - 1) / block_size_x,
                (N + block_size_y - 1) / block_size_y);
  // 按照x y z这个顺序排的
  dim3 block_dim(block_size_x / TN, block_size_y / TM);

  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  for (int i = 0; i < REPEAT_TIME; ++i)
    smem_gemm_blocktiling_2d_kernel<float><<<grid_dim, block_dim>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(), M, N, K);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "smem_gemm_blocktiling_2d Kernel execution time: "
            << milliseconds / REPEAT_TIME << " ms" << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return C;
}
